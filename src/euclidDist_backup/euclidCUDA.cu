#include "hip/hip_runtime.h"
/***********************************************
* # Copyright 2011. Thuy Diem Nguyen & Zejun Zheng
* # Contact: thuy1@e.ntu.edu.sg or zheng_zejun@sics.a-star.edu.sg
* #
* # GPL 3.0 applies.
* #
* ************************************************/

// Note: don't use_fast_math option

#include "euclidMain.h"
#include "euclidKernel.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>	

void writeVectorToFile_GPU(thrust::host_vector< thrust::pair<unsigned int, unsigned int> > h_pairVector, thrust::host_vector< float > h_distVector, string pairFileName, string distFileName, unsigned long long count, int fileId) {
	FILE * pairFile, * distFile; 
	string tempStr;	
	char buf[1000];
					
	sprintf(buf, "_%d", fileId);

	tempStr = pairFileName;
	tempStr.append(buf);
	pairFile = fopen(tempStr.c_str(), "wb");
	if (pairFile == NULL){
		printf("cannot open pairFile: %s\n", tempStr.c_str());
		exit(-1);
	}	
	tempStr = distFileName;
	tempStr.append(buf);
	distFile = fopen(tempStr.c_str(), "wb");
	if (distFile == NULL){
		printf("cannot open distFile: %s\n", tempStr.c_str());
		exit(-1);
	}
				
	thrust::device_vector<float> d_distVector = h_distVector; 
	thrust::device_vector< thrust::pair<unsigned int, unsigned int> > d_pairVector = h_pairVector;
	
	thrust::sort_by_key(d_distVector.begin(), d_distVector.end(), d_pairVector.begin());
				
	thrust::copy(d_distVector.begin(), d_distVector.end(), h_distVector.begin());
	thrust::copy(d_pairVector.begin(), d_pairVector.end(), h_pairVector.begin());
								
	int pairArray[BUF_SIZE*2];
	float distArray[BUF_SIZE];	

	int h = 0;
	thrust::pair<unsigned int, unsigned int> aPair;						
	
	cout << "write to : " << tempStr << " " << count << " pairs" << endl; 
				
	for (unsigned int i = 0; i < count; ++i)
	{					
		aPair = h_pairVector[i];	
		distArray[h] = h_distVector[i];
		pairArray[h*2] = aPair.first;
		pairArray[h*2+1] = aPair.second;		
		++h;		
		if (h == BUF_SIZE) {					
			fwrite(pairArray, sizeof(unsigned int), BUF_SIZE * 2, pairFile);		
			fwrite(distArray, sizeof(float), BUF_SIZE, distFile);		
			h = 0;
		}	
	}
	
	if (h > 0) {					
		fwrite(pairArray, sizeof(unsigned int), h * 2, pairFile);		
		fwrite(distArray, sizeof(float), h, distFile);
		h = 0;
	}	
		
	fclose(pairFile);
	fclose(distFile);				
}

void writeToVector(thrust::host_vector< thrust::pair<unsigned int, unsigned int> > & h_pairVector, thrust::host_vector< float > & h_distVector, float *h_distArray, int stageX, int stageY, int arrayDim, float threshold, unsigned long long & count) {

	int i, row, col, rowOffset, colOffset;	
	float dist;		
	int arraySize = arrayDim * arrayDim;
	
	rowOffset = stageX * arrayDim;
	colOffset = stageY * arrayDim;				

	// write result to output file
	for (i = 0; i < arraySize; ++i) 
	{
		row = rowOffset + (int)i / arrayDim;
		col = colOffset + (int)i % arrayDim;	
		dist = h_distArray[i];	
		if (dist < threshold || fabs(dist-threshold) < EPSILON)
		{
			h_pairVector[count] = thrust::make_pair(row, col);
			h_distVector[count] = dist;

			++count;
		}							
	}									
}

void computeEuclidDist_CUDA(float ** eReads, string pairFileName, string distFileName, int numReads, int numSeeds, float threshold, int arrayDim) {
	
	int i, j, stageX, stageY, row, offset, stageId;	
	unsigned long long totalNumPairs = 0, count = 0;
	int fileId = 0;
	
	int size = arrayDim * arrayDim;	
	int arraySize = size * NUM_STREAMS;
	int gridSize = (arrayDim + BLOCK_DIM - 1)/BLOCK_DIM;	
	int stageDim =  (numReads + arrayDim - 1)/arrayDim;
	
	// determine GRID_DIM and blockSize
	dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);	
	dim3 blocksPerGrid(gridSize, gridSize);	
		
	// get number of SMs on this GPU
	printf("size: %dx%d, arraySize: %d, stageDim: %dx%d\n", arrayDim, arrayDim, arraySize, stageDim, stageDim);	
	printf("blockSize: %dx%d, gridSize: %dx%d\n", BLOCK_DIM, BLOCK_DIM, gridSize, gridSize); 	

	// declare host variables
	float *inDistSet;
	inDistSet = (float*) malloc( numReads * numSeeds * sizeof(float) );
	
	for (i = 0; i < numReads; ++i)
	{
		row = i * numSeeds;
		for (j = 0; j < numSeeds; ++j)
			inDistSet[row + j] = eReads[i][j];	
	}

	for (i = 0; i < numReads; ++i)
		free(eReads[i]);
	free(eReads);			

	// declare device variables
	float *d_distArray;
	float *h_distArray;	

	checkCudaErrors( hipMalloc((void**)&d_distArray, arraySize * sizeof(float)) );	
	checkCudaErrors( hipHostMalloc((void**)&h_distArray, arraySize * sizeof(float)) );
			
	// use hipArray to store tupleArraySet
	hipChannelFormatDesc channelDesc=hipCreateChannelDesc<float>();
	hipArray *cuArray;
	size_t width, height;
	width = numSeeds*16;
	height = numReads/16;
	if ( (numReads&15) != 0) 
	 	++height;
	cout << "2D texture: width " << width << " height: " << height << endl;
	
	checkCudaErrors( hipMallocArray(&cuArray, &channelDesc, width, height) );
	checkCudaErrors( hipMemcpyToArray(cuArray, 0, 0, inDistSet, numSeeds * numReads * sizeof(float), hipMemcpyHostToDevice) );
	checkCudaErrors( hipBindTextureToArray(getTexRef(), cuArray, channelDesc) );
	free(inDistSet);			
		
	hipStream_t streams[NUM_STREAMS];

	for (i = 0; i < NUM_STREAMS; ++i) 
		checkCudaErrors( hipStreamCreate(&streams[i]) );			

	thrust::host_vector< float > h_distVector (MAX_NUM_PAIRS_GPU * 2);	
	thrust::host_vector< thrust::pair<unsigned int, unsigned int> > h_pairVector (MAX_NUM_PAIRS_GPU * 2);

	int stageSize = stageDim * (stageDim + 1) / 2;			
	for (j = 0; j < stageSize; j += NUM_STREAMS)
	{		

		for (i = 0; i < NUM_STREAMS; ++i) {
			offset = i * size;		
			stageId = i + j;
			
			if (stageId < stageSize) {
				Trag_reverse_eq(stageId, stageDim, stageX, stageY);													
						        								
				launchEuclidKernel(streams[i], blocksPerGrid, threadsPerBlock, d_distArray+offset, numReads, numSeeds, stageX, stageY, arrayDim);	
				
				checkCudaErrors( hipMemcpyAsync(h_distArray+offset, d_distArray+offset, size * sizeof(float), hipMemcpyDeviceToHost, streams[i]) );							
				 		
			}
		}		

		hipDeviceSynchronize();
		
		for (i = 0; i < NUM_STREAMS; ++i) {								
			offset = i * size;		
			stageId = i + j;				

			if (stageId < stageSize) {							
							
				Trag_reverse_eq(stageId, stageDim, stageX, stageY);		
				
				writeToVector(h_pairVector, h_distVector, h_distArray+offset, stageX, stageY, arrayDim, threshold, count);
																											
			}
		}				
		
		if (count >= MAX_NUM_PAIRS_GPU)
		{			
			h_pairVector.resize(count);
			h_distVector.resize(count);	
	
			writeVectorToFile_GPU(h_pairVector, h_distVector, pairFileName, distFileName, count, fileId);	
			
			h_pairVector.resize(MAX_NUM_PAIRS_GPU * 2);
			h_distVector.resize(MAX_NUM_PAIRS_GPU * 2);	

			++ fileId;
			totalNumPairs += count;
			count = 0;										
		}				
	}	
	
	if (count > 0)
	{
			h_pairVector.resize(count);
			h_distVector.resize(count);	
				
			writeVectorToFile_GPU(h_pairVector, h_distVector, pairFileName, distFileName, count, fileId);	
				
			totalNumPairs += count;										
	}	
	
	
	for (i = 0; i < NUM_STREAMS; ++i) 
		checkCudaErrors( hipStreamDestroy(streams[i]) );
				
	// clean up host variables	
	checkCudaErrors( hipHostFree(h_distArray) );
	checkCudaErrors( hipFree(d_distArray) );		
	
	// clean up device variables
	checkCudaErrors( hipUnbindTexture(getTexRef()) );
	checkCudaErrors( hipFreeArray(cuArray) );

	printf("totalNumPairs: %llu\n", totalNumPairs);	
}


